#include "hip/hip_runtime.h"
#include "variable_constraint_kernel.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "yuzu/foundation/memory/RelativePointer.hpp"
#include "yuzu/foundation/memory/pointer.hpp"
#include "yuzu/domain/boundary_conditions/BoundaryConditionData.hpp"
#include "yuzu/common/gpu.hpp"
#include "yuzu/utils/kernel_utils.hpp"
#include "stdio.h"

#define DOF_STATUS_FREE                         0

namespace ay = axis::yuzu;
namespace ayfm = axis::yuzu::foundation::memory;
namespace aydbc = axis::yuzu::domain::boundary_conditions;

struct VariableConstraintData
{
  const real *CurveDataPtr;
  real ScalingFactor;
  real ReleaseTime;
};

__global__ void __launch_bounds__(AXIS_YUZU_MAX_THREADS_PER_BLOCK)
  UpdateConstraintOnGPUKernel(uint64 numThreads, uint64 startIndex, 
  void *baseMemoryAddressOnGPU, real time, 
  axis::yuzu::foundation::memory::RelativePointer vectorMaskPtr)
{
  uint64 index = ay::GetThreadIndex(gridDim, blockIdx, blockDim, 
    threadIdx, startIndex);
  if (!ay::IsActiveThread(index, numThreads)) return;
  aydbc::BoundaryConditionData bcData(baseMemoryAddressOnGPU, index, sizeof(VariableConstraintData));
  real *bucket = bcData.GetOutputBucket();
  VariableConstraintData *data = 
    (VariableConstraintData *)bcData.GetCustomData();
  real curveCurVal = *data->CurveDataPtr;
  real scaleFactor = data->ScalingFactor;
  real releaseTime = data->ReleaseTime;
  uint64 dofId = bcData.GetDofId();
  *bucket = curveCurVal * scaleFactor;
  if (releaseTime >= 0 && time > releaseTime)
  {
    char *vectorMask = axis::yabsptr<char>(vectorMaskPtr);
    vectorMask[dofId] = DOF_STATUS_FREE;
  }
}

void axis::domain::boundary_conditions::UpdateConstraintOnGPU( 
  uint64 numThreadsToUse, uint64 startIndex, void *baseMemoryAddressOnGPU, 
  const axis::Dimension3D& gridDim, const axis::Dimension3D& blockDim, 
  void * streamPtr, real time, 
  axis::foundation::memory::RelativePointer vectorMaskPtr )
{
  dim3 grid, block;
  grid.x = gridDim.X; grid.y = gridDim.Y; grid.z = gridDim.Z;
  block.x = blockDim.X; block.y = blockDim.Y; block.z = blockDim.Z;
  UpdateConstraintOnGPUKernel<<<grid, block, 0, (hipStream_t)streamPtr>>>(
    numThreadsToUse, startIndex, baseMemoryAddressOnGPU, time,
    reinterpret_cast<ayfm::RelativePointer&>(vectorMaskPtr));
}
